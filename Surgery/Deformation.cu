#include "hip/hip_runtime.h"
//#include "math\CVector3d.h"//�������������

#include "Deformation.cuh"

#include <iostream>

const int MySIZE = 43250;

// error makro
#define CUERR {                                                              \
	hipError_t err;                                                         \
	if ((err = hipGetLastError()) != hipSuccess) {                         \
		std::cout << "CUDA error: " << hipGetErrorString(err) << " : "      \
					<< __FILE__ << ", line " << __LINE__ << std::endl;       \
		/*exit(1);                                                            */ \
	}                                                                        \
}

struct hipGraphicsResource *vertsPos_resource;

__device__ float3 d_pVelocities[MySIZE];


void InitVertsVelocities()
{
	float3 *d_pVel;
	
	hipGetSymbolAddress((void**)&d_pVel, d_pVelocities); CUERR
	hipMemset(d_pVel, 0, MySIZE * sizeof(float3));CUERR
}

extern "C" void RegisterCudaBuffer(struct hipGraphicsResource** dst, GLuint srcvbo)
{
	hipGraphicsGLRegisterBuffer(dst, srcvbo, cudaGraphicsMapFlagsNone); CUERR
}

size_t GetCudaMapPointer(struct hipGraphicsResource** res, void **ptr)
{
	size_t num_bytes;
	hipGraphicsMapResources(1, res, 0); CUERR
	hipGraphicsResourceGetMappedPointer((void **)ptr, &num_bytes, *res); CUERR

	return num_bytes;
}

__global__ void _OutputVertsPos(double* _poses)
{
	for (unsigned i = 0; i < 10; i++)
	{
		printf("%f %f %f\n", _poses[i * 3], _poses[i * 3 + 1], _poses[i * 3 + 2]);
	}
}

extern "C" void OutputVertsPos()
{
	double *poses;
	GetCudaMapPointer(&vertsPos_resource, (void**)&poses);

	_OutputVertsPos << <1, 1 >> > (poses);
	hipDeviceSynchronize(); CUERR
}