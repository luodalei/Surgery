#include "hip/hip_runtime.h"
//#include "math\CVector3d.h"//�������������

#include "Deformation.cuh"

#include <thrust\device_ptr.h>
#include <thrust\for_each.h>
#include <thrust\iterator\zip_iterator.h>
#include <thrust\sort.h>

#include <iostream>


// error makro
#define CUERR {                                                              \
	hipError_t err;                                                         \
	if ((err = hipGetLastError()) != hipSuccess) {                         \
		std::cout << "CUDA error: " << hipGetErrorString(err) << " : "      \
					<< __FILE__ << ", line " << __LINE__ << std::endl;       \
		/*exit(1);                                                            */ \
	}                                                                        \
}

//---------------------------------------------------------------------------
//global variables
//---------------------------------------------------------------------------
const int MySIZE = 43250;

// simulation parameters in constant memory
__constant__ PhysParam params;


//---------------------------------------------------------------------------
//function defination
//---------------------------------------------------------------------------

extern "C" void RegisterCudaBuffer(struct hipGraphicsResource** dst, GLuint srcvbo)
{
	hipGraphicsGLRegisterBuffer(dst, srcvbo, cudaGraphicsMapFlagsNone); CUERR
}

extern "C" void UnRegisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
	hipGraphicsUnregisterResource(cuda_vbo_resource);
}

extern "C" void AllocateArray(void **devPtr, size_t size)
{
	hipMalloc(devPtr, size); CUERR
}

extern "C" void FreeArray(void *devPtr)
{
	hipFree(devPtr); CUERR
}

extern "C" void CopyArrayToDevice(void *device, const void *host, int offset, int size)
{
	hipMemcpy((char*)device + offset, host, size, hipMemcpyHostToDevice); CUERR
}

extern "C" void CopyArrayFromDevice(void *host, const void *device, struct hipGraphicsResource **cuda_vbo_resource, int size)
{
	if (cuda_vbo_resource)
	{
		device = MapGLBufferObject(cuda_vbo_resource);
	}

	hipMemcpy(host, device, size, hipMemcpyDeviceToHost); CUERR

	if (cuda_vbo_resource)
	{
		UnMapGLBufferObject(*cuda_vbo_resource);
	}
}

extern "C" void CopyArrayFromDeviceToDevice(void *dTarg, void *dSrc, struct hipGraphicsResource **cuda_vbo_resource, int size)
{
	if (cuda_vbo_resource)
	{
		dSrc = MapGLBufferObject(cuda_vbo_resource);
	}

	hipMemcpy(dTarg, dSrc, size, hipMemcpyDeviceToDevice); CUERR

	if (cuda_vbo_resource)
	{
		UnMapGLBufferObject(*cuda_vbo_resource);
	}
}

extern "C" void SetValueToDevice(void *device, int value, int size)
{
	hipMemset(device, value, size); CUERR
}

extern "C" void* MapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
{
	void *ptr;
	hipGraphicsMapResources(1, cuda_vbo_resource, 0); CUERR
	
	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void**)&ptr, &num_bytes, *cuda_vbo_resource); CUERR

	return ptr;
}

extern "C" void UnMapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
	hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0); CUERR
}

extern "C" void SetParameters(PhysParam *hostParams)
{
	hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(PhysParam)); CUERR
}


//--------------------------------------------------------------------
//Find neighbors
//--------------------------------------------------------------------

// calculate position in uniform grid
__device__ int3 CalcGridPos(double3 p)
{
	int3 gridPos;
	gridPos.x = floor((p.x - params.worldOrigin.x) / params.cellSize.x);
	gridPos.y = floor((p.y - params.worldOrigin.y) / params.cellSize.y);
	gridPos.z = floor((p.z - params.worldOrigin.z) / params.cellSize.z);
	return gridPos;
}

// calculate address in grid from position (clamping to edges)
__device__ uint CalcGridHash(int3 gridPos)
{
	gridPos.x = gridPos.x & (params.gridSize.x - 1);  // wrap grid, assumes size is power of 2
	gridPos.y = gridPos.y & (params.gridSize.y - 1);
	gridPos.z = gridPos.z & (params.gridSize.z - 1);

	return gridPos.z * params.gridSize.y * params.gridSize.x + gridPos.y * params.gridSize.x + gridPos.x;
}

__global__ void CalcHashD(uint *gridParticleHash, uint *gridParticleIndex, double3 *pos, uint numParticles)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numParticles)
		return;

	volatile double3 p = pos[index];

	//get address in grid
	int3 gridPos = CalcGridPos(make_double3(p.x, p.y, p.z));
	uint hash = CalcGridHash(gridPos);

	gridParticleHash[index] = hash;
	gridParticleIndex[index] = index;
}


//Round a / b to nearest higher integer value
uint iDivUp(uint a, uint b)
{
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

extern "C" void ComputeGridSize(uint numParticles, uint defaultBlockSize, uint &numBlocks, uint &numThreads)
{
	numThreads = min(defaultBlockSize, numParticles);
	numThreads = nextPow2(numThreads);
	numBlocks = iDivUp(numParticles, numThreads);
}

extern "C" void CalcHash(uint *gridParticleHash, uint *gridParticleIndex, double3 *pos, int numParticles)
{
	uint numThreads, numBlocks;
	ComputeGridSize(numParticles, 256, numBlocks, numThreads);

	CalcHashD << <numBlocks, numThreads >> > (gridParticleHash, gridParticleIndex, pos, numParticles);
	hipDeviceSynchronize(); CUERR
}

// sort particles based on hash
extern "C" void SortParticles(uint *dGridParticleHash, uint *dGridParticleIndex, uint numParticles)
{
	thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash), thrust::device_ptr<uint>(dGridParticleHash + numParticles), thrust::device_ptr<uint>(dGridParticleIndex));
}

//------------------------------------------------------------------------------------------------------
// find the start & end particle index of each cell in the sorted hash array
//------------------------------------------------------------------------------------------------------
__global__ void FindCellStartEndD(uint   *cellStart,        // output: cell start index
								uint   *cellEnd,          // output: cell end index
								uint   *gridParticleHash, // input: sorted grid hashes
								uint    numParticles)
{
	extern __shared__ uint sharedHash[];    // blockSize + 1 elements

	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numParticles)
		return;

	uint hash;
	hash = gridParticleHash[index]; //��ǰparticle��cell����

	// Load hash data into shared memory so that we can look
	// at neighboring particle's hash value without loading
	// two hash values per thread
	sharedHash[threadIdx.x + 1] = hash;

	if (index > 0 && threadIdx.x == 0) //block's 1st thread except first block
	{
		// first thread in block must load neighbor particle hash
		sharedHash[0] = gridParticleHash[index - 1];
	}
	__syncthreads();

	// If this particle has a different cell index to the previous
	// particle then it must be the first particle in the cell,
	// As it isn't the first particle, it must also be the cell end of
	// the previous particle's cell (all particle's index in one cell < cell end)
	if (index == 0 || hash != sharedHash[threadIdx.x])
	{
		cellStart[hash] = index;

		if (index > 0)
			cellEnd[sharedHash[threadIdx.x]] = index;
	}
	if (index == numParticles - 1)
	{
		cellEnd[hash] = index + 1;
	}
}

extern "C" void FindCellStartEnd(uint *cellStart, uint *cellEnd,        //output
							uint *gridParticleHash, 
							uint numParticles, uint numCells)
{
	uint numThreads, numBlocks;
	ComputeGridSize(numParticles, 256, numBlocks, numThreads);

	// set all cells to empty
	hipMemset(cellStart, 0xffffffff, numCells * sizeof(uint)); CUERR

	uint smemSize = sizeof(uint) * (numThreads + 1);
	FindCellStartEndD << <numBlocks, numThreads, smemSize >> > (cellStart, cellEnd, gridParticleHash, numParticles);
	hipDeviceSynchronize(); CUERR
}

//----------------------------------------------------------------------------------------
//reorder pos & vel base on sorted particle indices
//----------------------------------------------------------------------------------------
__global__ void ReorderDataD(double3 *sortedPos,        // output: sorted positions
							double3 *sortedVel,        // output: sorted velocities
							uint   *gridParticleIndex, // input: sorted particle indices
							double3 *oldPos,           // input: unsorted position array
							double3 *oldVel,           // input: unsorted velocity array
							uint    numParticles)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numParticles)
		return;

	uint sortedIndex = gridParticleIndex[index];

	sortedPos[index] = oldPos[sortedIndex];
	sortedVel[index] = oldVel[sortedIndex];
}

extern "C" void ReorderData(double3 *sortedPos, double3 *sortedVel, //output
							uint *gridParticleIndex,
							double3 *oldPos, double3 *oldVel,
							uint numParticles)
{
	uint numThreads, numBlocks;
	ComputeGridSize(numParticles, 256, numBlocks, numThreads);

	ReorderDataD << <numBlocks, numThreads >> > (sortedPos, sortedVel, gridParticleIndex, oldPos, oldVel, numParticles);
	hipDeviceSynchronize(); CUERR
}


//-------------------------------------------------------------------------------
//find fixed distance neighbors from particles in neighboring cells
//-------------------------------------------------------------------------------
__device__ bool isTwoParticleWithinDst(double3 a, double3 b, double dst)
{
	double distance = (b.x - a.x) * (b.x - a.x) + (b.y - a.y) * (b.y - a.y) + (b.z - a.z) * (b.z - a.z);

	return distance <= (dst * dst) ? true : false;
}

__global__ void FindNeighborsWithinDstD(int* neighborIndex, //output
										uint *neighborCount, //output
										uint *gridParticleIndex,
										double3 *sortedPos, double dst,
										uint *cellStart, uint *cellEnd,
										uint numParticles)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numParticles)
		return;

	uint originalIndex = gridParticleIndex[index];
	double3 pos = sortedPos[index];
	int3 gridPos = CalcGridPos(pos);

	uint neighborId = 0;

	for (int z = -1; z <= 1; z++)
	{
		for (int y = -1; y <= 1; y++)
		{
			for (int x = -1; x <= 1; x++)
			{
				int3 neighbourGridPos = gridPos + make_int3(x, y, z);
				uint gridHash = CalcGridHash(neighbourGridPos);

				uint startIndex = cellStart[gridHash];
				if (startIndex != 0xffffffff) //cell is not empty
				{
					uint endIndex = cellEnd[gridHash];
					for (int j = startIndex; j < endIndex; j++)
					{
						if (j != index)
						{
							double3 targPos = sortedPos[j];
							if (isTwoParticleWithinDst(pos, targPos, dst))
							{
								if (neighborId >= 26)
									break;

								uint originalTargIndex = gridParticleIndex[j];

								neighborIndex[originalIndex * 26 + neighborId] = originalTargIndex; //Ĭ��ÿ��particle��Χ�����26���ھ�
								neighborId++;
							}
						}
					}
					neighborCount[originalIndex] = neighborId;
				}
			}
		}
	}
}

extern "C" void FindNeighborsWithinDst(int* neighborIndx, //output: original unsorted particles neighbor indeices
										uint *neighborCount, //output
										uint *gridParticleIndex,
										double3 *sortedPos, double dst,
										uint *cellStart, uint *cellEnd,
										uint numParticles)
{
	uint numThreads, numBlocks;
	ComputeGridSize(numParticles, 256, numBlocks, numThreads);

	FindNeighborsWithinDstD << <numBlocks, numThreads >> > (neighborIndx, neighborCount, gridParticleIndex, sortedPos, dst, cellStart, cellEnd, numParticles);
	hipDeviceSynchronize(); CUERR
}

//---------------------------------------------------------------------------
// build distance constraints based on neighborIdx
//---------------------------------------------------------------------------

__global__ void UpdateDistanceConstraintsD(int2 *dDistanceConstraints, //output
											double *dRestLength, //output
											uint *dNumDistanceConstraints, //output
											double3 *dPos,
											int *dNeighborIndex, uint *dNeighborCount,
											uint numParticles)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numParticles)
		return;

	uint constraintId = 0;
	for (uint j = 0; j < dNeighborCount[index]; j++)
	{
		int neighborIdx = dNeighborIndex[index * 26 + j];
		if (neighborIdx > index)
		{
			int2 pair = make_int2(index, neighborIdx);
			double3 dis = dPos[index] - dPos[neighborIdx];

			dDistanceConstraints[constraintId] = pair;
			dRestLength[constraintId] = YH::Length(dis);
			constraintId++;
		}
	}

	*dNumDistanceConstraints = constraintId;
}

extern "C" void UpdateDistanceConstraints(int2 *dDistanceConstraints, //output
											double *dRestLength, //output
											uint *dNumDistanceConstraints, //output
											double3 *dPos,
											int* dNeighborIndex, uint *dNeighborCount,
											uint numParticles)
{
	uint numThreads, numBlocks;
	ComputeGridSize(numParticles, 256, numBlocks, numThreads);

	UpdateDistanceConstraintsD << <1, 1 >> > (dDistanceConstraints, dRestLength, dNumDistanceConstraints, dPos, dNeighborIndex, dNeighborCount, numParticles);
	hipDeviceSynchronize(); CUERR
}

//---------------------------------------------------------------------------
// find nearby particle indices from target position
//---------------------------------------------------------------------------

__global__ void FindParticlesWithinDstFromTargetD(int* nearbyIndex, //output
													uint *nearbyCount, //output
													uint *gridParticleIndex,
													double3 *sortedPos,
													double3 targ, double dst,
													uint *cellStart, uint *cellEnd)
{
	int3 gridPos = CalcGridPos(targ);

	uint nearbyId = 0;

	//��dstȡ�ñȽϴ�ʱ��������Էſ�������Χ
	for (int z = -2; z <= 2; z++)
	{
		for (int y = -2; y <= 2; y++)
		{
			for (int x = -2; x <= 2; x++)
			{
				int3 neighbourGridPos = gridPos + make_int3(x, y, z);
				uint gridHash = CalcGridHash(neighbourGridPos);

				uint startIndex = cellStart[gridHash];
				if (startIndex != 0xffffffff) //cell is not empty
				{
					uint endIndex = cellEnd[gridHash];
					for (int j = startIndex; j < endIndex; j++)
					{
						double3 testPos = sortedPos[j];
						if (isTwoParticleWithinDst(testPos, targ, dst))
						{
							uint originalTestPosIndex = gridParticleIndex[j];
							nearbyIndex[nearbyId] = originalTestPosIndex;
							nearbyId++;
						}
					}
					nearbyCount[0] = nearbyId;
				}
			}
		}
	}
}

extern "C" void FindParticlesWithinDstFromTarget(int* nearbyIndex, //output
												uint *nearbyCount, //output
												uint *gridParticleIndex, 
												double3 *sortedPos, 
												double3 targ, double dst, 
												uint *cellStart, uint *cellEnd)
{
	//��ע�⣬���ﲻ��ÿ������ѭ������һ�飬Ҫ����
	FindParticlesWithinDstFromTargetD << <1, 1 >> > (nearbyIndex, nearbyCount, gridParticleIndex, sortedPos, targ, dst, cellStart, cellEnd);
	hipDeviceSynchronize(); CUERR
}

//-----------------------------------------------------------------------------
// update velocity: v += f/m * dt
//-----------------------------------------------------------------------------

__global__ void UpdataVelocitiesD(double3 *dVel, double dt, 
								int* nearbyIndex, uint *nearbyCount,
								double3 hapticForce, double3 environmentForce, 
								double *dInvMasses, 
								uint numParticles)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numParticles)
		return;

	for (uint i = 0; i < nearbyCount[0]; i++)
	{
		if (index == nearbyIndex[i])
		{
			dVel[index] += hapticForce * dInvMasses[index] * dt;
		}
	}

	dVel[index] += environmentForce * dInvMasses[index] * dt;
}

extern "C" void UpdateVelocities(double3 *dVel, double dt, 
								int* nearbyIndex, uint* nearbyCount, 
								double3 hapticForce, double3 environmentForce, 
								double *dInvMasses, 
								uint numParticles)
{
	uint numThreads, numBlocks;
	ComputeGridSize(numParticles, 256, numBlocks, numThreads);

	UpdataVelocitiesD << <numBlocks, numThreads >> > (dVel, dt, nearbyIndex, nearbyCount, hapticForce, environmentForce, dInvMasses, numParticles);
	hipDeviceSynchronize(); CUERR
}

//------------------------------------------------------------------------------
//semi implicit update vel & pos
//------------------------------------------------------------------------------

__global__ void SemiImplicitEulerD(double3 *dPredictPos, 
	double3 *dPos, double3 *dVel,
	double dt,
	int* nearbyIndex, uint *nearbyCount,
	double3 hapticForce, double3 environmentForce,
	double *dInvMasses,
	uint numParticles)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numParticles)
		return;

	dVel[index].x = dVel[index].y = dVel[index].z = 0;

	for (uint i = 0; i < nearbyCount[0]; i++)
	{
		if (index == nearbyIndex[i])
		{
			dVel[index] += hapticForce * dInvMasses[index] * dt;
		}
	}

	dVel[index] += environmentForce * dInvMasses[index] * dt;

	dPredictPos[index] = dPos[index] + dVel[index] * dt;
}

void SemiImplicitEuler(double3 *dPredictPos, 
	double3 *dPos, double3 *dVel,
	double dt, 
	int* nearbyIndex, uint *nearbyCount, 
	double3 hapticForce, double3 environmentForce, 
	double *dInvMasses, 
	uint numParticles)
{
	uint numThreads, numBlocks;
	ComputeGridSize(numParticles, 256, numBlocks, numThreads);

	SemiImplicitEulerD << <numBlocks, numThreads >> > (dPredictPos, dPos, dVel, dt, nearbyIndex, nearbyCount, hapticForce, environmentForce, dInvMasses, numParticles);
	hipDeviceSynchronize(); CUERR
}

//------------------------------------------------------------------------------
// predict position
//------------------------------------------------------------------------------

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T>
struct SharedMemory
{
	__device__ inline operator T *()
	{
		extern __shared__ int __smem[];
		return (T *)__smem;
	}

	__device__ inline operator const T *() const
	{
		extern __shared__ int __smem[];
		return (T *)__smem;
	}
};

template <class T>
__global__ void Reduce_ArraySumD(T* dOut, T* dIn, uint numData)
{
	uint globalId = blockIdx.x * blockDim.x + threadIdx.x;
	uint localId = threadIdx.x;

	if (globalId >= numData)
		return;

	T *sdata = SharedMemory<T>();

	sdata[localId] = dIn[globalId];
	__syncthreads();

	for (unsigned i = blockDim.x / 2; i > 0; i >>= 1)
	{
		if (localId < i)
		{
			sdata[localId] += sdata[localId + i];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (localId == 0)
	{
		dOut[blockIdx.x] = sdata[0];
	}
}


__global__ void AverageD(double3* dAver, double3* dSum, uint count)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index != 0)
		return;

	(*dAver).x = (*dSum).x / count;
	(*dAver).y = (*dSum).y / count;
	(*dAver).z = (*dSum).z / count;
}

extern "C" unsigned int nextPow2(unsigned int x)
{
	--x;
	x |= x >> 1;
	x |= x >> 2;
	x |= x >> 4;
	x |= x >> 8;
	x |= x >> 16;
	return ++x;
}

template<class T>
void SumFromArray(T *dOut, T *dSrc, uint numParticles)
{
	uint numThreads, numBlocks;
	ComputeGridSize(numParticles, 256, numBlocks, numThreads);

	T *dTmpSum, *dSum;
	AllocateArray((void**)&dTmpSum, sizeof(T) * numBlocks);
	AllocateArray((void**)&dSum, sizeof(T) * numBlocks);

	uint sMemSize = sizeof(T) * numThreads;
	Reduce_ArraySumD<T> << <numBlocks, numThreads, sMemSize >> > (dTmpSum, dSrc, numParticles); //��Լ�������뱣֤�߳�����2��ָ����
	hipDeviceSynchronize(); CUERR

	while (numBlocks != 1)
	{
		ComputeGridSize(numBlocks, 256, numBlocks, numThreads);

		sMemSize = sizeof(T) * numThreads;
		Reduce_ArraySumD<T> << <numBlocks, numThreads, sMemSize >> > (dSum, dTmpSum, numParticles);
		hipDeviceSynchronize(); CUERR

		CopyArrayFromDeviceToDevice(dTmpSum, dSum, 0, sizeof(T) * numBlocks);
	}

	CopyArrayFromDeviceToDevice(dOut, dSum, 0, sizeof(T));
}

template 
void SumFromArray<double3>(double3 *dOut, double3 *dSrc, uint numParticles);

template
void SumFromArray<Mat3d>(Mat3d *dOut, Mat3d *dSrc, uint numParticles);


__global__ void Prepare_L_I_R_ArrayD(double3 *dLArray, Mat3d *dIArray, double3 *dR, double3 *dPos, double3 *dVel, double3 *dAverPos, uint numParticles)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numParticles)
		return;

	double3 ri = dPos[index] - dAverPos[0];
	dLArray[index] = YH::Cross(ri, dVel[index]);

	Mat3d tmp(0, ri.z, -ri.y,
			-ri.z, 0, ri.x,
			ri.y, -ri.x, 0);
	dIArray[index] = tmp * YH::Transpose(tmp);

	dR[index] = ri;
}

__global__ void PredictPosD(double3 *dPredictPos, //output
							double3 *dPos, 
							double3 *dVel, double3 *dAverVel, 
							Mat3d *dI, double3 *dL, double3 *dR, 
							double damping, double dt, 
							uint numParticles)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numParticles)
		return;

	double3 W = YH::Invert(dI[0]) * dL[0];
	double3 deltVi = dAverVel[0] + YH::Cross(W, dR[index]) - dVel[index];

	dVel[index] += damping * deltVi;
	dPredictPos[index] = dPos[index] + dVel[index] * dt;
}

extern "C" void IntegrateExplicitWithDamping(double3 *dPredictPos, double3 *dPos, double3 *dVel, double damping, double dt, uint numParticles)
{
	double3 *dSumPos, *dSumVel, *dAverPos, *dAverVel;
	AllocateArray((void**)&dSumPos, sizeof(double3));
	AllocateArray((void**)&dSumVel, sizeof(double3));
	AllocateArray((void**)&dAverPos, sizeof(double3));
	AllocateArray((void**)&dAverVel, sizeof(double3));

	//---------------------λ��ƽ��----------------------
	SumFromArray<double3>(dSumPos, dPos, numParticles);
	AverageD << <1, 1 >> > (dAverPos, dSumPos, numParticles);
	//-------------------------�ٶ�ƽ��------------------
	SumFromArray<double3>(dSumVel, dVel, numParticles);
	AverageD << <1, 1 >> > (dAverVel, dSumVel, numParticles);

	double3 hSumPos;
	CopyArrayFromDevice(&hSumPos, dSumPos, 0, sizeof(double3));
	//printf("dSumPos: %f %f %f\n", hSumPos.x, hSumPos.y, hSumPos.z);
	
	//----------------------Ԥ��λ��_׼��---------------------
	Mat3d hI, *dI; 
	hI.Identity();

	double3 *dL, *dLArray;
	double3 *dR;
	Mat3d *dIArray;

	AllocateArray((void**)&dI, sizeof(Mat3d));
	AllocateArray((void**)&dL, sizeof(double3));
	AllocateArray((void**)&dLArray, sizeof(double3) * numParticles);
	AllocateArray((void**)&dR, sizeof(double3) * numParticles);
	AllocateArray((void**)&dIArray, sizeof(Mat3d) * numParticles);

	CopyArrayToDevice(dI, &hI, 0, sizeof(Mat3d));
	SetValueToDevice(dL, 0, sizeof(double3));
	SetValueToDevice(dLArray, 0, sizeof(double3) * numParticles);
	SetValueToDevice(dR, 0, sizeof(double3) * numParticles);
	SetValueToDevice(dIArray, 0, sizeof(Mat3d) * numParticles);

	uint numThreads, numBlocks;
	ComputeGridSize(numParticles, 256, numBlocks, numThreads);
	Prepare_L_I_R_ArrayD << <numBlocks, numThreads >> > (dLArray, dIArray, dR, dPos, dVel, dAverPos, numParticles);

	//��һ�ι�Լ
	SumFromArray<double3>(dL, dLArray, numParticles);
	SumFromArray<Mat3d>(dI, dIArray, numParticles);

	//test output
	/*CopyArrayFromDevice(&hI, dI, 0, sizeof(Mat3d));

	printf("dI: \n%f %f %f \n %f %f %f \n %f %f %f\n", hI(0, 0), hI(0, 1), hI(0, 2),
													hI(1, 0), hI(1, 1), hI(1, 2),
													hI(2, 0), hI(2, 1), hI(2, 2));
	*/
	//-------------------------Ԥ��λ��-----------------------------
	PredictPosD << <numBlocks, numThreads >> > (dPredictPos, dPos, dVel, dAverVel, dI, dL, dR, damping, dt, numParticles);
	hipDeviceSynchronize(); CUERR
	

	FreeArray(dSumPos);
	FreeArray(dSumVel);
	FreeArray(dAverPos);
	FreeArray(dAverVel);
	FreeArray(dI);
	FreeArray(dL);
	FreeArray(dR);
	FreeArray(dLArray);
	FreeArray(dIArray);
}

//-----------------------------------------------------------------------------
// update all constraints
//-----------------------------------------------------------------------------

__global__ void UpdateDistanceConstraintsD(double3 *dPredictPos, 
	double* dInvMasses,
	int2 *dDistanceConstraint, double compressionStiffness, double stretchStiffness, double *dRestLength, int numDistanceConstraints)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numDistanceConstraints)
		return;

	int p0 = dDistanceConstraint[index].x;
	int p1 = dDistanceConstraint[index].y;
	double restLen = dRestLength[index];

	double invMass0 = dInvMasses[p0];
	double invMass1 = dInvMasses[p1];
	double invMassSum = invMass0 + invMass1;

	if (invMassSum <= 0.000001)
		return;

	double3 n = dPredictPos[p1] - dPredictPos[p0];
	double len = YH::Length(n);
	n = YH::Normalize(n);

	double3 corr;
	if (len < restLen)
		corr = compressionStiffness * n * (len - restLen) / invMassSum;
	else
		corr = stretchStiffness * n * (len - restLen) / invMassSum;

	dPredictPos[p0] -= invMass0 * corr;
	dPredictPos[p1] += invMass1 * corr;
}

void UpdateAllConstraints(double3 *dPredictPos, //output
	double *dInvMasses,
	int2 *dDistanceConstraint, double compressionStiffness, double stretchStiffness, double *dRestLength, int numDistanceConstraints,
	uint numIteration)
{
	uint numThreads0, numBlocks0;
	ComputeGridSize(numDistanceConstraints, 256, numBlocks0, numThreads0);

	for (uint i = 0; i < numIteration; i++)
	{
		UpdateDistanceConstraintsD << <numBlocks0, numThreads0 >> > (dPredictPos, dInvMasses, dDistanceConstraint, compressionStiffness, stretchStiffness, dRestLength, numDistanceConstraints);
	}
	hipDeviceSynchronize(); CUERR
}

//---------------------------------------------------------------------------
// Determine final vel
//---------------------------------------------------------------------------

__global__ void DetermineFinalVelD(double3 *dVel, //output
	double3 *dPredictPos, double3 *dPos, double dt, uint numParticles)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numParticles)
		return;

	dVel[index] = (dPredictPos[index] - dPos[index]) / dt;
}

void DetermineFinalVel(double3 *dPredictPos, double3 *dPos, double3 *dVel, double dt, uint numParticles)
{
	uint numThreads, numBlocks;
	ComputeGridSize(numParticles, 256, numBlocks, numThreads);

	DetermineFinalVelD << <numBlocks, numThreads >> > (dVel, dPredictPos, dPos, dt, numParticles);
	hipDeviceSynchronize(); CUERR
}